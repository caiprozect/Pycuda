
#include <hip/hip_runtime.h>
__global__ void mapToNumb(
	const int N, //Number of whole threads
	const int M, //Length of subseq that one thread handles
	char* seq, 
	int* numb_seq
)
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int idx = gid * M;
	int i, letter;

	if(idx < N*M) {
	for(i=0; i < M; i++) {
		letter = seq[idx+i];
		if(letter == 'A') {
			numb_seq[idx+i] = 0;
		} else {
		if(letter == 'C') {
			numb_seq[idx+i] = 1;
		} else {
		if(letter == 'G') {
			numb_seq[idx+i] = 2;
		} else {
		if(letter == 'U') {
			numb_seq[idx+i] = 3;
		} else {
			numb_seq[idx+i] = (-1) * (int)(powf(4, (float)3));
		}
		}
		}
		}
	}
	}
}

__global__ void genNumbCodon(
	const int N,
	const int M,
	int* numb_seq,
	int* codon_seq
	)
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int idx = gid * M;
	int i, k;
	int codon_numb, loc_idx, numb, base;

	for(i=0; i < M; i++) {
		codon_numb = 0;
		loc_idx = idx + i;
		if(loc_idx <= N*M -3 + 1) {
			for(k=0; k<3; k++) {
				numb = numb_seq[loc_idx];
				base = (int)powf(4, (float)(2-k));
				codon_numb += numb * base;
			}
			codon_seq[loc_idx] = codon_numb;
		}
	}	
}

__global__ void mapToAA(
	const int N,
	const int M,
	char* rna_codon_tab,
	int* codon_seq,
	char* aa_seq
	)
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	int idx = gid * M;
	int codon_idx, loc_idx;
	int i;
	
	for(i=0; i < M; i++) {
		loc_idx = idx + i;
		codon_idx = codon_seq[loc_idx];
		if(loc_idx <= N*M -3 + 1) {
			if(codon_idx >= 0) {
				aa_seq[loc_idx] = rna_codon_tab[codon_idx];
			}
		}
	}

}















